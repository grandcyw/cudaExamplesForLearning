
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void add_atomic(int *a)
{
    int local=__ldg(a);
    printf("Thread %d read value: %d\n", threadIdx.x, local);
    atomicAdd(a,1);
    // __syncthreads();
}

int main()
{
    int *a;
    hipMallocManaged((void**)&a,sizeof(int));
    *a=4;
    add_atomic<<<1,2>>>(a);
    hipError_t error=hipGetLastError();
    if(error)
    {
        printf("Error: %s\n", hipGetErrorString(error));
    }
    hipDeviceSynchronize();
    printf("Result: %d\n", *a);
    hipFree(a);
    return 0;
}